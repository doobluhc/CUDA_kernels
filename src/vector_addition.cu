#include <hip/hip_runtime.h>

__global__ void vector_add(const float* d_input1, const float* d_input2, float* d_output, size_t n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n){
        d_output[i] = d_input1[i] + d_input2[i];
    }
}

extern "C" void solution(const float* d_input1, const float* d_input2, float* d_output, size_t n) {    
    int threadsPerBlock = 32;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_input1, d_input2, d_output, n);
    hipDeviceSynchronize();
}